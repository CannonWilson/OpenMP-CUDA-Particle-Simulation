#include "hip/hip_runtime.h"
// Imports
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <random>
#include <chrono>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "particle_functions.h"


// Provide some namespace shortcuts for timing
using std::cout;
using std::chrono::high_resolution_clock;
using std::chrono::duration;

const float G = 6.67430;//e-11;  // Gravitational constant
const float particle_mass = 1;
const bool saveDensities = true; // Should the simulation save densities (true) or Positions (false)
const char* FILE_NAME = "gpu_results_12-11.txt";


// CUDA kernel for handling collisions
__global__ void handle_collisions_kernel(float* positions, float* velocities, float* accelerations, int num_particles) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < num_particles) {
        // Check and handle collisions for the x dimension
        if (positions[tid * 3] < 0.0f) {
            positions[tid * 3] = 0.0f;
            velocities[tid * 3] = 0.0f;
            accelerations[tid * 3] = 0.0f;
        } else if (positions[tid * 3] > 100.0f) {
            positions[tid * 3] = 100.0f;
            velocities[tid * 3] = 0.0f;
            accelerations[tid * 3] = 0.0f;
        }

        // Check and handle collisions for the y dimension
        if (positions[tid * 3 + 1] < 0.0f) {
            positions[tid * 3 + 1] = 0.0f;
            velocities[tid * 3 + 1] = 0.0f;
            accelerations[tid * 3 + 1] = 0.0f;
        } else if (positions[tid * 3 + 1] > 100.0f) {
            positions[tid * 3 + 1] = 100.0f;
            velocities[tid * 3 + 1] = 0.0f;
            accelerations[tid * 3 + 1] = 0.0f;
        }

        // Check and handle collisions for the z dimension
        if (positions[tid * 3 + 2] < 0.0f) {
            positions[tid * 3 + 2] = 0.0f;
            velocities[tid * 3 + 2] = 0.0f;
            accelerations[tid * 3 + 2] = 0.0f;
        } else if (positions[tid * 3 + 2] > 100.0f) {
            positions[tid * 3 + 2] = 100.0f;
            velocities[tid * 3 + 2] = 0.0f;
            accelerations[tid * 3 + 2] = 0.0f;
        }
    }
}

// CUDA kernel for frame update
__global__ void frame_update_kernel(float* positions, float* velocities, float* accelerations, int num_particles) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < num_particles) {
        float acc_x = 0.0f;
        float acc_y = 0.0f;
        float acc_z = 0.0f;

        for (int j = 0; j < num_particles; ++j) {
            if (tid != j) {
                float dx = positions[j * 3 + 0] - positions[tid * 3 + 0];
                float dy = positions[j * 3 + 1] - positions[tid * 3 + 1];
                float dz = positions[j * 3 + 2] - positions[tid * 3 + 2];
                float r = sqrtf(dx * dx + dy * dy + dz * dz);

                if (r > 0 && !std::isnan(r)) {
                    float force = (G * particle_mass * particle_mass) / (r * r);

                    acc_x += force * (dx / r);
                    acc_y += force * (dy / r);
                    acc_z += force * (dz / r);
                }
            }
        }

        // Update accelerations
        accelerations[tid * 3 + 0] = acc_x;
        accelerations[tid * 3 + 1] = acc_y;
        accelerations[tid * 3 + 2] = acc_z;

        // Update velocities and positions
        velocities[tid * 3 + 0] += accelerations[tid * 3 + 0];
        velocities[tid * 3 + 1] += accelerations[tid * 3 + 1];
        velocities[tid * 3 + 2] += accelerations[tid * 3 + 2];

        positions[tid * 3 + 0] += velocities[tid * 3 + 0];
        positions[tid * 3 + 1] += velocities[tid * 3 + 1];
        positions[tid * 3 + 2] += velocities[tid * 3 + 2];
    }
}

// Wrapper function for handling collisions on GPU
void handle_collisions_cuda(float* positions, float* velocities, float* accelerations, int num_particles) {
    int num_threads = 256;
    int num_blocks = (num_particles + num_threads - 1) / num_threads;

    handle_collisions_kernel<<<num_blocks, num_threads>>>(positions, velocities, accelerations, num_particles);
    hipDeviceSynchronize();  // Ensure the kernel execution is completed before proceeding
}

// Wrapper function for frame update on GPU
void frame_update_cuda(float* positions, float* velocities, float* accelerations, int num_particles) {
    int num_threads = 256;
    int num_blocks = (num_particles + num_threads - 1) / num_threads;

    frame_update_kernel<<<num_blocks, num_threads>>>(positions, velocities, accelerations, num_particles);
    hipDeviceSynchronize();  // Ensure the kernel execution is completed before proceeding

    // Handle the collisions next
    handle_collisions_cuda(positions, velocities, accelerations, num_particles);
}





// Main
int main(int argc, char* argv[]) {

    // Check len of command-line args
    if (argc != 3) {
        std::cout << "Check # of command-line args.";
        return 1; // fail
    }

    // Read command-line arguments
    int num_frames = atoi(argv[1]);
    int num_particles = atoi(argv[2]);

    // Check command-line args are valid
    if (num_frames < 1 || num_particles < 1) {
        std::cout << "Check values for command line args.";
        return 1; // fail
    }

    // Simulator constants
    int box_width = 100;
    int v_init = 0; // Assume velocity will be initialized to 0 in all 3 directions


    // Other useful vars derived from above constants
    int box_volume = box_width * box_width * box_width;
    int particles_per_dim = static_cast<int>(std::round(std::pow(num_particles, 1.0 / 3.0)));
    float grid_spacing = static_cast<float>(box_width) / particles_per_dim; // Distance between particles in each direction
    
    // Random number setup
    float min = 0, max = static_cast<float>(box_width);
    std::random_device rd;
    std::mt19937 eng(rd());
    std::uniform_real_distribution<float> distr(min, max);


    // Timing setup
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    duration<double, std::milli> duration_sec;
    
    // Files info
    FILE* file_ptr;
    file_ptr = fopen(FILE_NAME, "w");


    // Initialize arrays using CUDA managed memory
    float* positions;
    float* velocities;
    float* accelerations;

    hipMallocManaged(&positions, num_particles * 3 * sizeof(float));
    hipMallocManaged(&velocities, num_particles * 3 * sizeof(float));
    hipMallocManaged(&accelerations, num_particles * 3 * sizeof(float));

    // Populate positions with randomly distributed particles
   for (int i=0; i<num_particles; i++) {
        positions[i*3] = distr(eng);
        positions[i*3 + 1] = distr(eng);
        positions[i*3 + 2] = distr(eng);
   }

    // Start timer
    hipEventRecord(start);

    // Print out initial positions before running sim
    // if (!saveDensities) {
    //     print_positions(positions, num_particles, file_ptr);
    // }

    // Run the sim
    for (int i=0; i<num_frames-1; i++) {
        frame_update_cuda(positions, velocities, accelerations, num_particles);
        // if (saveDensities) {
        //     print_densities(positions, num_particles, file_ptr, particles_per_dim);
        // }
        // else {
        //     print_positions(positions, num_particles, file_ptr);
        // }
    }

    // Stop timer and print timing result
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);
    std::cout << "Time taken: " << elapsed_time << "ms";

    // Don't forget to free the allocated memory
    hipFree(positions);
    hipFree(velocities);
    hipFree(accelerations);

    // Close opened file
    fclose(file_ptr);

    return 0;
}